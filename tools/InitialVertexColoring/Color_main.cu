#include "hip/hip_runtime.h"
#include <stdio.h>
#include "all_structure_undir.cuh"
#include "gpuFunctions_undir.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include<vector>
#include <chrono>
#include <algorithm>
#include "compactor.cuh"
#include "supportingFunctions.cu"


#define THREADS_PER_BLOCK 1024 //we can change it

using namespace std;
using namespace std::chrono;



/*
1st arg: original graph file name
2nd arg: no. of nodes
****main commands to run****
nvcc -o op_main Color_main.cu
./op_main original_graph_file_name number_of_nodes
*/
int main(int argc, char* argv[]) {

	int nodes, edges, deviceId, numberOfSMs;
	hipError_t cudaStatus;
	nodes = atoi(argv[2]);
	edges = 0; //for inital graph we considering we have 0 edge
	//char* inputColorfile = argv[4];
	int totalInsertion = 0;
	bool zeroDelFlag = false, zeroInsFlag = false;
	vector<ColList> AdjList; //stores input graph in 2D adjacency list
	vector<ColWt> AdjListFull; //Row-major implementation of adjacency list (1D)
	ColWt* AdjListFull_device; //1D array in GPU to store Row-major implementation of adjacency list 
	int* AdjListTracker_device; //1D array to track offset for each node's adjacency list
	vector<changeEdge> allChange_Ins, allChange_Del;
	changeEdge* allChange_Ins_device; //stores all change edges marked for insertion in GPU
	changeEdge* allChange_Del_device; //stores all change edges marked for deletion in GPU
	int* counter;
	int* affected_marked;
	int* affectedNodeList;
	int* vertexcolor;
	int* previosVertexcolor;
	float total_time = 0.0;
	int SCmaskArrayElement;


	////Get gpu device id and number of SMs
	hipGetDevice(&deviceId);
	hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
	size_t  numberOfBlocks = 32 * numberOfSMs;


	////read input vertex color label
	int maxColor = -1;
	cudaStatus = hipMallocManaged(&vertexcolor, nodes * sizeof(int));
	cudaStatus = hipMemset (vertexcolor, 0, nodes * sizeof(int) ); //set initial color to 0
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at vertexcolor structure");
	}
	//commenting below part as this code is for initial coloring
	//read_Input_Color(vertexcolor, inputColorfile, &maxColor);
	//SCmaskArrayElement = maxColor / 32 + 2; //we take 1 element extra to manage the situation if number of color increases
	
	SCmaskArrayElement = 3; //reduce or increase it if required -> also change maskArray size accordingly in GPU functions
	//printf("Max color id in input graph: %d\n", maxColor);


	////Read Original input graph
	AdjList.resize(nodes);
	int* AdjListTracker = (int*)malloc((nodes + 1) * sizeof(int));//we take nodes +1 to store the start ptr of the first row
	//read_graphEdges(AdjList, argv[1]); //commenting as considering at initial graph we have no edge
	/*printf("printing adj list:\n");
	for (int i = 0; i < nodes; i++)
	{
		printf("\nAdj list for %d:\n", i);
		for (int j = 0; j < AdjList[i].size(); j++)
		{
			printf("%d:", AdjList[i][j].col);
		}
	}*/



	////Read change edges input
	readin_changes(argv[1], allChange_Ins, allChange_Del, AdjList, totalInsertion, vertexcolor); //here we consider the actual graph edges as inserted edges.
	int totalChangeEdges_Ins = allChange_Ins.size();
	if (totalChangeEdges_Ins == 0) {
		zeroInsFlag = true;
	}
	int totalChangeEdges_Del = allChange_Del.size();
	if (totalChangeEdges_Del == 0) {
		zeroDelFlag = true;
	}
	/*for (int i = 0; i < totalChangeEdges_Ins; i++)
	{
		printf("\nEffective Change edges INS: %d %d:\n", allChange_Ins[i].node1, allChange_Ins[i].node2);
	}*/


	////Transfer input graph, changed edges to GPU and set memory advices
	transfer_data_to_GPU(AdjList, AdjListTracker, AdjListFull, AdjListFull_device,
		nodes, edges, totalInsertion, AdjListTracker_device, zeroInsFlag,
		allChange_Ins, allChange_Ins_device, totalChangeEdges_Ins,
		deviceId, totalChangeEdges_Del, zeroDelFlag, allChange_Del_device,
		counter, affected_marked, affectedNodeList, previosVertexcolor,/*updatedAffectedNodeList_del, updated_counter_del,*/ allChange_Del, numberOfBlocks);


	//test code
	//printf("printing AdjListFull_device:\n"); //after adding ins edges and deleting(or setting flag =-1) del edges
	//printAdj << <numberOfBlocks, THREADS_PER_BLOCK >> > (AdjListFull_device, AdjListTracker_device, nodes);
	//hipDeviceSynchronize();

	//Saturation color mask
	//int* saturationColorMask;
	//cudaStatus = hipMallocManaged(&saturationColorMask, nodes * sizeof(int));
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMalloc failed at saturationColor structure");
	//}
	////compute saturation mask
	//computeSCMask << <numberOfBlocks, THREADS_PER_BLOCK >> > (AdjListFull_device, AdjListTracker_device, nodes, saturationColorMask, vertexcolor);
	//hipDeviceSynchronize();

	//printMask << <numberOfBlocks, THREADS_PER_BLOCK >> > (saturationColorMask, AdjListTracker_device, nodes);
	//hipDeviceSynchronize();
	//test code ends


	//input color validation //it gives error as we add ins edges while reading
	/*validate << < numberOfBlocks, THREADS_PER_BLOCK >> > (AdjListFull_device, AdjListTracker_device, nodes, vertexcolor);
	hipDeviceSynchronize();*/


	////Initialize supporting variables
	int* change = 0;
	cudaStatus = hipMallocManaged(&change, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at change structure");
	}



	////test::For random number generation in CUDA kernel 
	/*hiprandState* states;
	hipMalloc((void**)&states, numberOfBlocks * THREADS_PER_BLOCK * sizeof(hiprandState));*/









	////process change edges////

	////Process del edges
	if (zeroDelFlag != true) {
		auto startTimeDelEdge = high_resolution_clock::now(); //Time calculation start

		deleteEdge << < numberOfBlocks, THREADS_PER_BLOCK >> > (/*states,*/ allChange_Del_device, vertexcolor, previosVertexcolor, totalChangeEdges_Del, AdjListFull_device, AdjListTracker_device, affected_marked, change, SCmaskArrayElement);
		hipDeviceSynchronize(); //comment this if required

		auto stopTimeDelEdge = high_resolution_clock::now();//Time calculation ends
		auto durationDelEdge = duration_cast<microseconds>(stopTimeDelEdge - startTimeDelEdge);// duration calculation
		cout << "**Time taken for processing del edges: "
			<< float(durationDelEdge.count()) / 1000 << " milliseconds**" << endl;
		total_time += float(durationDelEdge.count()) / 1000;
	}

	//Process ins edges
	if (zeroInsFlag != true) {
		auto startTimeInsEdge = high_resolution_clock::now(); //Time calculation start

		insEdge << < numberOfBlocks, THREADS_PER_BLOCK >> > (allChange_Ins_device, vertexcolor, previosVertexcolor, totalChangeEdges_Ins, AdjListFull_device, AdjListTracker_device, affected_marked, change, SCmaskArrayElement);
		hipDeviceSynchronize(); //comment this if required

		auto stopTimeInsEdge = high_resolution_clock::now();//Time calculation ends
		auto durationInsEdge = duration_cast<microseconds>(stopTimeInsEdge - startTimeInsEdge);// duration calculation
		cout << "**Time taken for processing ins edges: "
			<< float(durationInsEdge.count()) / 1000 << " milliseconds**" << endl;
		total_time += float(durationInsEdge.count()) / 1000;
	}

	if (zeroDelFlag != true) {
		hipFree(allChange_Del_device);
	}
	if (zeroInsFlag != true) {
		hipFree(allChange_Ins_device);
	}


	auto startTimeDelNeig = high_resolution_clock::now(); //Time calculation start

	//we use compactor in place of just adding directly using atomic fn to avoid duplication of affected vertices in list
	*counter = cuCompactor::compact<int, int>(affected_marked, affectedNodeList, nodes, predicate(), THREADS_PER_BLOCK);
	//recolor affected neighbors
	while (*change > 0)
	{
		*change = 0;
		//reset affected_del to 0
		hipMemset(affected_marked, 0, nodes * sizeof(int));
		//printf("after memset 0: affected_del flag for %d = %d \n", 1, affected_del[1]);

		//find eligible neighbors which should be updated
		findEligibleNeighbors << < numberOfBlocks, THREADS_PER_BLOCK >> > (affectedNodeList, AdjListFull_device, AdjListTracker_device, affected_marked, previosVertexcolor, vertexcolor, counter);

		//find the next frontier: it collects the vertices to be recolored and store without duplicate in affectedNodeList
		*counter = cuCompactor::compact<int, int>(affected_marked, affectedNodeList, nodes, predicate(), THREADS_PER_BLOCK);
		/*printf("After findEligibleNeighbors: affectedNodeList_del elements:\n");
		for (int i = 0; i < *counter_del; i++)
		{
			printf("%d:", affectedNodeList_del[i]);
		}*/
		hipMemset(affected_marked, 0, nodes * sizeof(int)); //new
		//recolor the eligible neighbors
		recolorNeighbor << < numberOfBlocks, THREADS_PER_BLOCK >> > (affectedNodeList, vertexcolor, previosVertexcolor, AdjListFull_device, AdjListTracker_device, affected_marked, counter, change, SCmaskArrayElement);
		hipDeviceSynchronize();
	}
	auto stopTimeDelNeig = high_resolution_clock::now();//Time calculation ends
	auto durationDelNeig = duration_cast<microseconds>(stopTimeDelNeig - startTimeDelNeig);// duration calculation
	cout << "**Time taken for processing affected neighbors: "
		<< float(durationDelNeig.count()) / 1000 << " milliseconds**" << endl;
	total_time += float(durationDelNeig.count()) / 1000;
	cout << "****Total Time for Vertex Color Update: "
		<< total_time << " milliseconds****" << endl;

	////print output vertex color
	//printf("\nprinting output vertex colors:\n");
	//for (int i = 0; i < nodes; i++)
	//{
	//	printf("%d:%d\n", i, vertexcolor[i]);
	//}
	
	//Print max color id used
	maxColor = -1;
	for (int i = 0; i < nodes; i++)
	{
		if (vertexcolor[i] > maxColor) {
			maxColor = vertexcolor[i];
		}
	}
	printf("highest color id used: %d\n", maxColor);

	validate << < numberOfBlocks, THREADS_PER_BLOCK >> > (AdjListFull_device, AdjListTracker_device, nodes, vertexcolor);
	hipDeviceSynchronize();

	//print all color
	for (int i = 0; i < nodes; i++)
	{
		printf("%d %d\n", i, vertexcolor[i]);
	}

	
	hipFree(change);
	hipFree(vertexcolor);
	hipFree(affected_marked);
	hipFree(affectedNodeList);
	hipFree(counter);
	hipFree(AdjListFull_device);
	hipFree(AdjListTracker_device);
	hipFree(previosVertexcolor);
	return 0;
}